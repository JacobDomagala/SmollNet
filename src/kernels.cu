#include "hip/hip_runtime.h"
#include "helpers.hpp"
#include "kernels.cuh"

#include <cstdio>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace smollnet {
__device__ __forceinline__ void compute_dimensions(int (&dims)[3], size_t idx,
                                                   const StrideInfo &s) {

  if (s.rank == 3) {
    int64_t rest = s.output_size[1] * s.output_size[2];
    dims[0] = idx / rest;
    int64_t rem = idx % rest;
    dims[1] = rem / s.output_size[2];
    dims[2] = rem % s.output_size[2];
  } else if (s.rank == 2) {
    dims[0] = idx / s.output_size[1];
    dims[1] = idx % s.output_size[1];
    dims[2] = 0;
  } else { // rank == 1
    dims[0] = idx;
    dims[1] = 0;
    dims[2] = 0;
  }
}

__global__ void random_init(float *out, size_t total, size_t seed) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= total)
    return;

  hiprandState state;
  hiprand_init(seed, idx, 0, &state);

  out[idx] = hiprand_uniform(&state);
}

void launch_random_init(void *out, size_t total) {
  dim3 block(256);
  dim3 grid((total + block.x - 1) / block.x);
  unsigned long long seed = time(nullptr);

  random_init<<<grid, block>>>(static_cast<float *>(out), total, seed);

  CHECK_CUDA(hipGetLastError());
}

__global__ void negative_kernel(float *ptr, size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < total)
    ptr[idx] *= -1.0f;
}

void launch_negative(void *ptr, size_t total) {
  dim3 block = 256;
  dim3 grid = (block.x + total - 1) / block.x;

  negative_kernel<<<grid, block>>>(static_cast<float *>(ptr), total);
}

template <typename T> __global__ void fill_kernel(T *data, size_t n, T value) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    data[idx] = value;
}

void launch_fill(float *ptr, size_t numElems, float val) {
  dim3 block(256);
  dim3 grid((numElems + block.x - 1) / block.x);
  fill_kernel<<<grid, block>>>(ptr, numElems, val);
  CHECK_CUDA(hipGetLastError());
}

template <typename T>
__global__ void add_kernel(T *out, T *left, T *right, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    out[idx] = left[idx] + right[idx];
}

void launch_add(float *out, float *left, float *right, size_t numElems) {
  dim3 block(256);
  dim3 grid((numElems + block.x - 1) / block.x);
  add_kernel<<<grid, block>>>(out, left, right, numElems);
  CHECK_CUDA(hipGetLastError());
}

__global__ void add_strided_kernel(float *__restrict__ out,
                                   const float *__restrict__ a,
                                   const float *__restrict__ b, StrideInfo s,
                                   size_t total) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= total)
    return;

  // Decode linear index -> (i,j,k)
  int dims[3] = {0, 0, 0};
  compute_dimensions(dims, idx, s);

  int64_t offA = dims[0] * s.a_stride[0] + dims[1] * s.a_stride[1] +
                 dims[2] * s.a_stride[2];
  int64_t offB = dims[0] * s.b_stride[0] + dims[1] * s.b_stride[1] +
                 dims[2] * s.b_stride[2];

  out[idx] = a[offA] + b[offB];
}

void launch_add_strided(void *dst, void *a, void *b, const StrideInfo &s,
                        size_t total) {
  dim3 blk(256);
  dim3 grd((total + blk.x - 1) / blk.x);

  add_strided_kernel<<<grd, blk>>>(static_cast<float *>(dst),
                                   static_cast<const float *>(a),
                                   static_cast<const float *>(b), s, total);
  CHECK_CUDA(hipGetLastError());
}

template <typename T>
__global__ void mul_kernel(T *out, T *left, T scalar, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    out[idx] = left[idx] * scalar;
}

template <typename T>
__global__ void mul_kernel(T *out, T *left, T *right, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    out[idx] = left[idx] * right[idx];
}

void launch_mul(float *out, float *left, float *right, size_t numElems) {
  dim3 block(256);
  dim3 grid((numElems + block.x - 1) / block.x);
  mul_kernel<<<grid, block>>>(out, left, right, numElems);
  CHECK_CUDA(hipGetLastError());
}

__global__ void mul_strided_kernel(float *__restrict__ out,
                                   const float *__restrict__ a,
                                   const float *__restrict__ b, StrideInfo s,
                                   size_t total) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= total)
    return;

  // Decode linear index -> (i,j,k)
  int dims[3] = {0, 0, 0};
  compute_dimensions(dims, idx, s);

  int64_t offA = dims[0] * s.a_stride[0] + dims[1] * s.a_stride[1] +
                 dims[2] * s.a_stride[2];
  int64_t offB = dims[0] * s.b_stride[0] + dims[1] * s.b_stride[1] +
                 dims[2] * s.b_stride[2];

  out[idx] = a[offA] * b[offB];
}

void launch_mul_strided(void *dst, void *a, void *b, const StrideInfo &s,
                        size_t total) {
  dim3 blk(256);
  dim3 grd((total + blk.x - 1) / blk.x);

  mul_strided_kernel<<<grd, blk>>>(static_cast<float *>(dst),
                                   static_cast<const float *>(a),
                                   static_cast<const float *>(b), s, total);
  CHECK_CUDA(hipGetLastError());
}

template <typename T>
__global__ void sub_kernel(T *out, T *left, T *right, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    out[idx] = left[idx] - right[idx];
}

void launch_sub(float *out, float *a, float *b, size_t numElems) {
  dim3 block(256);
  dim3 grid((numElems + block.x - 1) / block.x);
  sub_kernel<<<grid, block>>>(out, a, b, numElems);
  CHECK_CUDA(hipGetLastError());
}

__global__ void sub_strided_kernel(float *out, float *a, float *b, StrideInfo s,
                                   size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= total)
    return;

  // Decode linear index -> (i,j,k)
  int dims[3] = {0, 0, 0};
  compute_dimensions(dims, idx, s);

  int64_t offA = dims[0] * s.a_stride[0] + dims[1] * s.a_stride[1] +
                 dims[2] * s.a_stride[2];
  int64_t offB = dims[0] * s.b_stride[0] + dims[1] * s.b_stride[1] +
                 dims[2] * s.b_stride[2];

  out[idx] = a[offA] - b[offB];
}
void launch_sub_strided(void *out, void *a, void *b, const StrideInfo &s,
                        size_t total) {
  dim3 block = 256;
  dim3 grid = (total + block.x - 1) / block.x;
  sub_strided_kernel<<<grid, block>>>(static_cast<float *>(out),
                                      static_cast<float *>(a),
                                      static_cast<float *>(b), s, total);
}

template <int AXIS>
__global__ void sum_kernel(const float *__restrict__ in,
                           float *__restrict__ out, int64_t d0, int64_t d1,
                           int64_t d2) {
  const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Remaining elements -> each elem will compute
  const int64_t num_elems = AXIS == 0 ? d1 * d2 : AXIS == 1 ? d0 * d2 : d0 * d1;

  // const int64_t num_elems = d0 * d1 * d2;

  extern __shared__ float sMem[];

  if (idx >= num_elems)
    return;

  int64_t stride = 0;
  int64_t i0 = 0;
  int64_t i1 = 0;
  int64_t i2 = 0;
  int64_t base = 0;
  int64_t size = 0;

  if constexpr (AXIS == 0) {
    stride = d1 * d2;
    i0 = idx / stride;
    i1 = idx / d2;
    i2 = idx % d2;
    size = d0;
    base = stride;
  } else if constexpr (AXIS == 1) {

  } else {
  }

  const float *p = in + idx;
  float acc = 0.f;

#pragma unroll
  for (int i = 0; i < size; ++i) {
    acc += *p;
    p += stride;
  }

  out[idx] = acc;
}

template <int AXIS>
__global__ void sum_kernel_atomic(const float *__restrict__ in,
                                  float *__restrict__ out, int64_t d0,
                                  int64_t d1, int64_t d2) {
  const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Remaining elements -> each elem will compute
  // const int64_t num_elems = AXIS == 0 ? d1 * d2 : AXIS == 1 ? d0 * d2 : d0 *
  // d1;

  const int64_t num_elems = d0 * d1 * d2;

  extern __shared__ float sMem[];

  if (idx >= num_elems)
    return;

  int64_t stride = 0;
  int64_t i0 = 0;
  int64_t i1 = 0;
  int64_t i2 = 0;
  int64_t base = 0;
  int64_t size = 0;

  if constexpr (AXIS == 0) {
    stride = d1 * d2;
    i0 = idx / stride;
    i1 = (idx / d2) % d1;
    i2 = idx % d2;
    size = d0;
    base = stride;
  } else if constexpr (AXIS == 1) {

  } else {
  }

  const float *p = in + idx;

  atomicAdd(out + (i1 * d2 + i2), *p);
}

template <unsigned int BLOCK_SIZE>
__global__ void reduction_kernel(const float *__restrict__ in,
                                 float *__restrict__ out, const int64_t d0,
                                 const int64_t d1, const int64_t d2) {
  // Each block handles one output element at index (i1, i2)
  // The grid should be launched as dim3(d2, d1)
  const int64_t i1 = blockIdx.y;
  const int64_t i2 = blockIdx.x;

  // The output memory location for this block
  const int64_t out_idx = i1 * d2 + i2;

  // This thread's partial sum
  float my_sum = 0.0f;

  // Stride between consecutive elements along the reduction axis (d0)
  const int64_t reduction_stride = d1 * d2;

  // Each thread in the block cooperates to reduce the slice.
  // We start at an offset corresponding to this thread's lane in the block.
  // The loop strides by the block size, so the block as a whole sweeps
  // through the entire slice along the d0 axis.
  for (int64_t i = threadIdx.x; i < d0; i += BLOCK_SIZE) {
    // *** CORRECT COALESCED READ ***
    // Consecutive threads (threadIdx.x, threadIdx.x+1, ...) access
    // consecutive memory locations because 'out_idx' is constant for the block
    // and 'i' increases by 1 for each thread.
    my_sum += in[out_idx + i * reduction_stride];
  }

  // --- In-block reduction using shared memory ---
  extern __shared__ float sMem[];
  sMem[threadIdx.x] = my_sum;
  __syncthreads();

  // Standard parallel reduction within the shared memory tile
  for (unsigned int s = BLOCK_SIZE / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      sMem[threadIdx.x] += sMem[threadIdx.x + s];
    }
    __syncthreads();
  }

  // Thread 0 of the block writes the final, reduced sum for this slice.
  // No atomics are needed because each block writes to a unique location.
  if (threadIdx.x == 0) {
    out[out_idx] = sMem[0];
  }
}

// __global__ void
// tiled_sum_2d(const float* in, float* out, size_t d0, size_t d1, size_t d2) {

//   const int64_t idx = threadIdx.x + blockDim.x * blockIdx.x
//   // 16x16 tile and also block
//   extern __shared__ float sMem[];

//   size_t stride = 0;

//   // transposed manner so warp can local reduce
//   sMem[threadIdx.y + threadIdx.x * blockDim.y] = in[threadIdx.x + threadIdx.y
//   * stride];

//   __syncthreads();

//   float v = 0.0f;
//   #pragma unroll
//   for (int off = 16; off > 0; off >>= 1)
//     v += __shfl_down_sync(0xffffffff, v, off);

//   if ((threadIdx.x & 31) == 0)
//     atomicAdd(out + offset, v);
// }

template <typename T, unsigned int BLOCK_SIZE = 256, unsigned int VEC = 4>
__global__ void reduce_axis0_kernel(const T *__restrict__ in,
                                    T *__restrict__ out, size_t N, size_t M) {
  const size_t colStart = (blockIdx.x * BLOCK_SIZE + threadIdx.x) * VEC;
  if (colStart >= M)
    return;

  T accum[VEC];
#pragma unroll
  for (int v = 0; v < VEC; ++v)
    accum[v] = T(0);

  for (size_t row = 0; row < N; ++row) {
    const size_t base = row * M + colStart;
#pragma unroll
    for (int v = 0; v < VEC; ++v) {
      const size_t col = colStart + v;
      if (col < M)
        accum[v] += in[base + v];
    }
  }

#pragma unroll
  for (int v = 0; v < VEC; ++v) {
    const size_t col = colStart + v;
    if (col < M)
      out[col] = accum[v];
  }
}

__global__ void warp_level_sum(const float *__restrict__ in,
                               float *__restrict__ out, size_t dim_size,
                               size_t offset,
                               size_t n) // n = d0*d1*d2
{
  // We always launch this kernel as 1D block, the only difference can be a grid
  // dim
  int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t i = blockIdx.y;
  int64_t d = blockIdx.z;

  int64_t idx = d * offset + i * dim_size + j;
  int64_t out_idx = d * offset + i;

  float v = (idx < n and j < dim_size) ? in[idx] : 0.0f;

  extern __shared__ float sMem[];

#pragma unroll
  for (int off = 16; off > 0; off >>= 1)
    v += __shfl_down_sync(0xffffffff, v, off);

  if ((threadIdx.x & 31) == 0 and j < dim_size) {
    sMem[threadIdx.x / 32] = v;
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    float acc = 0.0f;
    for (int i = 0; i < blockDim.x / 32; ++i) {
      acc += sMem[i];
    }
    atomicAdd(out + out_idx, acc);
  }
}

void launch_sum_dim0(void *out, void *in, int64_t d0, int64_t d1, int64_t d2) {

  // Contigious memory access -> warp level reduce!
  if (d1 * d2 == 1) {
    dim3 block = 256;
    const auto total = d0 * d1 * d2;
    dim3 grid = (total + 256 - 1) / 256;
    warp_level_sum<<<grid, block, (256 / 32) * sizeof(float)>>>(
        static_cast<const float *>(in), static_cast<float *>(out), d0, 0,
        total);
  } else {
    // For strided pattern use tile + smem
  }
}

void launch_sum_dim1(void *out, void *in, int64_t d0, int64_t d1, int64_t d2) {
  // Contigious memory access -> warp level reduce!
  if (d2 == 1) {
    dim3 block(256, 1, 1);
    const auto total = d0 * d1 * d2;
    dim3 grid((d1 + block.x - 1) / block.x, d0, 1);
    warp_level_sum<<<grid, block, (256 / 32) * sizeof(float)>>>(
        static_cast<const float *>(in), static_cast<float *>(out), d1, 0,
        total);
  } else {
    // For strided pattern use tile + smem
    int block = 256;
    int grid = (d1 + block - 1) / block;
    sum_kernel<1><<<grid, block, d1 * sizeof(float)>>>(
        static_cast<const float *>(in), static_cast<float *>(out), d0, d1, d2);
  }

  CHECK_CUDA(hipGetLastError());
}

void launch_sum_dim2(void *out, void *in, int64_t d0, int64_t d1, int64_t d2) {
  // Since we only support Rank3 Tensors, this is always contigious memory
  dim3 block(256, 1, 1);
  const auto total = d0 * d1 * d2;
  dim3 grid((block.x + d2 - 1) / block.x, d1, d0);

  fmt::print(
      "Launching warp_level_sum<<<({},{},{}), ({},{},{})>>>(in,out,{},{},{})\n",
      grid.x, grid.y, grid.z, block.x, block.y, block.z, d2, d0 * d1, total);
  warp_level_sum<<<grid, block, (256 / 32) * sizeof(float)>>>(
      static_cast<const float *>(in), static_cast<float *>(out), d2, d1, total);

  CHECK_CUDA(hipGetLastError());
}

__global__ void matmul_kernel(float *__restrict__ C,
                              const float *__restrict__ A,
                              const float *__restrict__ B,
                              const StrideInfo strides, const SizeInfo sizes,
                              const int tile_width) {
  const int col = blockIdx.x * blockDim.x + threadIdx.x; // N‑index
  const int row = blockIdx.y * blockDim.y + threadIdx.y; // M‑index

  const int M = strides.output_size[0];
  const int N = strides.output_size[1];
  const int K = sizes.a_size[1]; // = sizes.b_size[0]

  const bool in_bounds = (row < M) && (col < N);

  extern __shared__ float s_mem[];
  float *As = s_mem;                           // tile from A (M×K)
  float *Bs = s_mem + tile_width * tile_width; // tile from B (K×N)

  float acc = 0.0f;
  const int num_tiles = (K + tile_width - 1) / tile_width;

  for (int t = 0; t < num_tiles; ++t) {
    const int a_col = t * tile_width + threadIdx.x; // K‑index into A
    const int b_row = t * tile_width + threadIdx.y; // K‑index into B

    // Load current tiles into shared memory, zero‑padding out‑of‑range
    // elements.
    As[threadIdx.y * tile_width + threadIdx.x] =
        (row < M && a_col < K) ? A[row * K + a_col] : 0.0f;

    Bs[threadIdx.y * tile_width + threadIdx.x] =
        (b_row < K && col < N) ? B[b_row * N + col] : 0.0f;

    __syncthreads();

    // Multiply–accumulate over the valid fragment length.
    const int elems = min(tile_width, K - t * tile_width);
#pragma unroll
    for (int e = 0; e < elems; ++e)
      acc +=
          As[threadIdx.y * tile_width + e] * Bs[e * tile_width + threadIdx.x];

    __syncthreads();
  }

  if (in_bounds)
    C[row * N + col] = acc;
}

void launch_matmul(void *out, void *left, void *right,
                   const StrideInfo &strides, const SizeInfo &sizes,
                   size_t total) {
  constexpr int TILE = 16;
  dim3 block(TILE, TILE);

  const int M = strides.output_size[0]; // rows of C
  const int N = strides.output_size[1]; // cols of C

  dim3 grid((N + TILE - 1) / TILE,  // x‑dim ← N
            (M + TILE - 1) / TILE); // y‑dim ← M

  size_t smem_bytes = 2 * TILE * TILE * sizeof(float);

  matmul_kernel<<<grid, block, smem_bytes>>>(
      static_cast<float *>(out), static_cast<const float *>(left),
      static_cast<const float *>(right), strides, sizes, TILE);

  CHECK_CUDA(hipGetLastError());
}

__global__ void relu_kernel(float *out, float *in, size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total)
    out[idx] = fmaxf(in[idx], 0.0f);
}

void launch_relu(void *out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  relu_kernel<<<grid, block>>>(static_cast<float *>(out),
                               static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void relu_grad_kernel(float *out, float *grad_out, float *in,
                                 size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total)
    out[idx] = (in[idx] > 0.0f) ? grad_out[idx] : 0.0f;
}

void launch_relu_grad(void *out, void *grad_out, void *in, size_t total) {
  int block = 256;
  int grid = (total + block - 1) / block;

  relu_grad_kernel<<<grid, block>>>(static_cast<float *>(out),
                                    static_cast<float *>(grad_out),
                                    static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void gelu_kernel(float *out, float *in, size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total) {
    constexpr float sqrt_2_over_pi = 0.7978845608f;
    out[idx] =
        0.5f * in[idx] *
        (1.0f + tanhf(sqrt_2_over_pi *
                      (in[idx] + 0.044715f * in[idx] * in[idx] * in[idx])));
  }
}

void launch_gelu(void *out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  gelu_kernel<<<grid, block>>>(static_cast<float *>(out),
                               static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void gelu_grad_kernel(float *out, float *grad_out, float *in,
                                 size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total) {
    // const float a = std::sqrt(2.0f / M_PI);
    constexpr float a = 0.7978845608f;
    constexpr float b = 0.044715f;
    float x3 = in[idx] * in[idx] * in[idx];
    float h = in[idx] + b * x3;
    float tanh_ax = tanhf(a * h);
    float sech2 = 1.0f - tanh_ax * tanh_ax;
    float h_prime = 1.0f + 3.0f * b * in[idx] * in[idx];

    float g = 0.5f * (1.0f + tanh_ax) + 0.5f * in[idx] * sech2 * a * h_prime;
    out[idx] = grad_out[idx] * g;
  }
}

void launch_gelu_grad(void *out, void *grad_out, void *in, size_t total) {
  int block = 256;
  int grid = (total + block - 1) / block;

  gelu_grad_kernel<<<grid, block>>>(static_cast<float *>(out),
                                    static_cast<float *>(grad_out),
                                    static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void tanh_kernel(float *out, float *in, size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total)
    out[idx] = tanhf(in[idx]);
}

void launch_tanh(void *out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  tanh_kernel<<<grid, block>>>(static_cast<float *>(out),
                               static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void tanh_grad_kernel(float *out, float *grad_out, float *in,
                                 size_t total) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < total)
    out[idx] = grad_out[idx] * (1.0f - in[idx] * in[idx]);
}

void launch_tanh_grad(void *out, void *grad_out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  tanh_grad_kernel<<<grid, block>>>(static_cast<float *>(out),
                                    static_cast<float *>(grad_out),
                                    static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void sigmoid_kernel(float *output, float *input, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] =
        1.0f / (1.0f + expf(-input[idx])); // Apply sigmoid to each element
  }
}

void launch_sigmoid(void *out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  sigmoid_kernel<<<grid, block>>>(static_cast<float *>(out),
                                  static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void sigmoid_grad_kernel(float *output, float *grad_out,
                                    float *input, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] = grad_out[idx] * input[idx] * (1.0f - input[idx]);
  }
}

void launch_sigmoid_grad(void *out, void *grad_out, void *in, size_t total) {

  int block = 256;
  int grid = (total + block - 1) / block;

  sigmoid_grad_kernel<<<grid, block>>>(static_cast<float *>(out),
                                       static_cast<float *>(grad_out),
                                       static_cast<float *>(in), total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void mse_kernel(float *out, const float *__restrict__ pred,
                           const float *__restrict__ target, std::size_t n) {
  extern __shared__ float sdata[];
  std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t stride = blockDim.x * gridDim.x;
  float local_sum = 0.0f;

  for (; idx < n; idx += stride) {
    float diff = pred[idx] - target[idx];
    local_sum += diff * diff;
  }

  sdata[threadIdx.x] = local_sum;
  __syncthreads();

  // reduction in shared memory
  for (std::size_t offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset)
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    __syncthreads();
  }

  if (threadIdx.x == 0)
    atomicAdd(out, sdata[0]);
}

void launch_mse(void *out, void *pred, void *target, size_t total) {
  int block = 256;
  int grid = (total + block - 1) / block;

  mse_kernel<<<grid, block, block * sizeof(float)>>>(
      static_cast<float *>(out), static_cast<float *>(pred),
      static_cast<float *>(target), total);

  mul_kernel<<<1, 1>>>(static_cast<float *>(out), static_cast<float *>(out),
                       1.0f / static_cast<float>(total), 1);
  CHECK_CUDA(hipGetLastError());
}
__global__ void sgd_kernel(float *w, const float *grad, float lr,
                           size_t total) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < total) {
    w[idx] -= lr * grad[idx];
  }
}

void launch_sgd_update(void *p, void *g, float lr, size_t total) {
  dim3 block = 256;
  dim3 grid = (total + block.x - 1) / block.x;
  sgd_kernel<<<grid, block>>>(static_cast<float *>(p),
                              static_cast<const float *>(g), lr, total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void mse_grad_kernel(float *g, const float *p, const float *t,
                                float coeff, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n)
    g[idx] = coeff * (p[idx] - t[idx]);
}

void launch_mse_grad(void *grad, void *pred, void *target, float coeff,
                     size_t total) {
  int block = 256;
  int grid = (total + block - 1) / block;
  mse_grad_kernel<<<grid, block>>>(static_cast<float *>(grad),
                                   static_cast<float *>(pred),
                                   static_cast<float *>(target), coeff, total);
  CHECK_CUDA(hipGetLastError());
}

__global__ void variance_step1_kernel(float *out, float *in, float *mean,
                                      const size_t batch_size,
                                      const size_t num_features) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= batch_size * num_features)
    return;

  int batch_num = idx / num_features;
  out[idx] = powf(mean[batch_num] - in[idx], 2);
}

__global__ void variance_step2_kernel(float *out, float *in,
                                      const size_t batch_size,
                                      const size_t num_features) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= batch_size)
    return;

  float acc = 0.0f;

  for (int i = 0; i < num_features; ++i) {
    acc += in[idx * num_features + i];
  }

  acc /= num_features;

  out[idx] = acc;
}

void launch_variance(void *variance, void *staging_buffer, void *in, void *mean,
                     size_t batch_size, size_t num_features) {
  dim3 block = 256;
  dim3 grid = (block.x + batch_size * num_features - 1) / block.x;

  variance_step1_kernel<<<grid, block>>>(
      static_cast<float *>(staging_buffer), static_cast<float *>(in),
      static_cast<float *>(mean), batch_size, num_features);

  variance_step2_kernel<<<grid, block>>>(static_cast<float *>(variance),
                                         static_cast<float *>(staging_buffer),
                                         batch_size, num_features);
}

__global__ void mean_2d_kernel(float *out, float *in, size_t d0, size_t d1) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= d0)
    return;

  float acc = 0.0f;
  for (int i = 0; i < d1; ++i) {
    acc += in[idx * d1 + i];
  }

  acc /= d1;

  out[idx] = acc;
}

void launch_mean_2d(void *out, void *in, size_t d0, size_t d1) {
  dim3 block = 256;
  dim3 grid = (block.x + d0 * d1 - 1) / block.x;

  mean_2d_kernel<<<grid, block>>>(static_cast<float *>(out),
                                  static_cast<float *>(in), d0, d1);
}

__global__ void layer_norm_kernel(float *out, float *features, float *mean,
                                  float *variance, float *gamma, float *beta,
                                  size_t batch_size, size_t num_features) {
  auto idx = threadIdx.x + blockDim.x * blockIdx.x;
  const auto total = batch_size * num_features;

  if (idx >= total)
    return;

  int batch_num = idx / num_features;

  constexpr float epsilon = 1e-5f;
  float normalized =
      (features[idx] - mean[batch_num]) / sqrtf(variance[batch_num] + epsilon);

  out[idx] = gamma[batch_num] * normalized + beta[batch_num];
}

void launch_layer_norm(void *out, void *features, void *mean, void *variance,
                       void *gamma, void *beta, size_t batch_size,
                       size_t num_features) {
  dim3 block = 256;
  size_t total = batch_size * num_features;
  dim3 grid = (block.x + total - 1) / block.x;

  layer_norm_kernel<<<grid, block>>>(
      static_cast<float *>(out), static_cast<float *>(features),
      static_cast<float *>(mean), static_cast<float *>(variance),
      static_cast<float *>(gamma), static_cast<float *>(beta), batch_size,
      num_features);
}

__global__ void layer_norm_grad_kernel(float *out_grad,
                                       const float *normalized_input,
                                       const float *scaled_gradient,
                                       const float *variance,
                                       const float *summed_scale,
                                       const float *summed_scaled_input,
                                       size_t batch_size, size_t num_features) {
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
  const size_t total = batch_size * num_features;
  if (idx >= total)
    return;

  const size_t row = idx / num_features;

  constexpr float eps = 1e-5f;
  const float inv_std = rsqrtf(variance[row] + eps); // per-sample variance
  const float m1 = summed_scale[row] / num_features; // Σδ / D
  const float m2 = summed_scaled_input[row] / num_features; // Σδ·ẋ / D

  const float hat_x = normalized_input[idx];
  const float delta = scaled_gradient[idx]; // δ = dy * γ

  const float res = inv_std * (delta - m1 - hat_x * m2); // ∂L/∂x
  out_grad[idx] = res;
}

void launch_layer_norm_grad(void *out, void *normalized_input,
                            void *scaled_gradient, void *variance,
                            void *summed_scale, void *summed_scaled_input,
                            size_t batch_size, size_t num_features) {

  dim3 block = 256;
  size_t total = batch_size * num_features;
  dim3 grid = (block.x + total - 1) / block.x;
  layer_norm_grad_kernel<<<grid, block>>>(
      static_cast<float *>(out), static_cast<float *>(normalized_input),
      static_cast<float *>(scaled_gradient), static_cast<float *>(variance),
      static_cast<float *>(summed_scale),
      static_cast<float *>(summed_scaled_input), batch_size, num_features);
}

} // namespace smollnet
